#include "hip/hip_runtime.h"
#include "systeme.h"
#include <ctype.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define BLOCK_SIZE 256

void systeme_init(systeme_t *s) {
    s->m = NULL;
    s->matrice = NULL;
    s->n = 0;
}

void systeme_lecture(systeme_t *s, char* pathFile) {
    printf("Lecture du fichier %s\n", pathFile);
    FILE *fichier = fopen(pathFile, "r");
    if (fichier == NULL) {
        fprintf(stderr, "Erreur: impossible d'ouvrir le fichier\n");
        exit(1);
    }

    int temp = fscanf(fichier, "%d", &s->n);
    s->m = (double *)malloc((s->n * (s->n + 1)) * sizeof(double));
    if (s->m == NULL) {
        fprintf(stderr, "Erreur: impossible d'allouer la mémoire\n");
        exit(1);
    }

    s->matrice = (double **)malloc(s->n * sizeof(double *));
    if (s->matrice == NULL) {
        fprintf(stderr, "Erreur lors de l'allocation de mémoire pour matrice\n");
        exit(1);
    }

    for (int i = 0; i < s->n; i++) {
        s->matrice[i] = &(s->m[i * (s->n + 1)]);
    }

    for (int i = 0; i < s->n; i++) {
        for (int j = 0; j < (s->n + 1); j++) {
         int temp =  fscanf(fichier, "%lf", &s->matrice[i][j]);
        }
    }

    fclose(fichier);
}

void systeme_afficher(systeme_t s) {
    printf("======Affichage du système:======\n");
    for (int i = 0; i < s.n; i++) {
        for (int j = 0; j < s.n + 1; j++) {
            printf("%lf ", s.matrice[i][j]);
        }
        printf("\n");
    }
}

__device__ void systeme_echanger_lignes(double* d_matrice, int n, int iter, int i) {
    double tmp;
    for (int j = 0; j < n + 1; j++) {
        tmp = d_matrice[iter * (n + 1) + j];
        d_matrice[iter * (n + 1) + j] = d_matrice[i * (n + 1) + j];
        d_matrice[i * (n + 1) + j] = tmp;
    }
}

__global__ void solve_system(double* d_matrice, int n) {
    extern __shared__ double shared[];

    for (int iter = 0; iter < n - 1; iter++) {
        int idx = threadIdx.x + blockIdx.x * blockDim.x;

        // Sélection du pivot
        double max_val = fabs(d_matrice[iter * (n + 1) + iter]);
        int pivot_row = iter;
        for (int i = iter + 1; i < n; i++) {
            double val = fabs(d_matrice[i * (n + 1) + iter]);
            if (val > max_val) {
                max_val = val;
                pivot_row = i;
            }
        }

        // Échange des lignes si nécessaire
        if (pivot_row != iter) {
            if (idx == 0) {
                systeme_echanger_lignes(d_matrice, n, iter, pivot_row);
            }
            __syncthreads();
        }

//         // afficher matrice
//         if (idx == 0) {
//             printf("Matrice après échange de lignes:\n");
//             for (int i = 0; i < n; i++) {
//                 for (int j = 0; j < n + 1; j++) {
//                     printf("%lf ", d_matrice[i * (n + 1) + j]);
//                 }
//                 printf("\n");
//             }
//             printf("\n");
//         }

        // Copie la ligne pivot dans la mémoire partagée
        if (threadIdx.x < n + 1 && idx < n+1) {
            shared[threadIdx.x] = d_matrice[iter * (n + 1) + threadIdx.x];
        }
        __syncthreads();

        // Normalise la ligne pivot
        double pivot = shared[iter];
        if (pivot == 0.0) continue; // Évite la division par zéro
        if (threadIdx.x < n + 1) {
            shared[threadIdx.x] /= pivot;
            d_matrice[iter * (n + 1) + threadIdx.x] = shared[threadIdx.x];
        }
        __syncthreads();

        // Soustraction des lignes
        if (idx >= iter + 1 && idx < n) {
            double coef = d_matrice[idx * (n + 1) + iter];
            for (int j = iter; j < n + 1; j++) {
                d_matrice[idx * (n + 1) + j] -= coef * shared[j];
            }
        }
        __syncthreads();

//         // afficher matrice
//         if (idx == 0) {
//             printf("Matrice après soustraction des lignes:\n");
//             for (int i = 0; i < n; i++) {
//                 for (int j = 0; j < n + 1; j++) {
//                     printf("%lf ", d_matrice[i * (n + 1) + j]);
//                 }
//                 printf("\n");
//             }
//             printf("\n");
//         }



        // Copie les résultats de la mémoire partagée vers la matrice
        if (threadIdx.x < n + 1 && idx < n) {
            d_matrice[iter * (n + 1) + threadIdx.x] = shared[threadIdx.x];
        }
        __syncthreads();
    }
}

void systeme_resoudre(systeme_t *s, int boolAffichage) {
    int size = s->n * (s->n + 1) * sizeof(double);
    double* d_matrice;
    int nbBlocks = (s->n + BLOCK_SIZE - 1) / BLOCK_SIZE;
    size_t sharedSize = (s->n + 1) * sizeof(double);

    // Copie les données vers le GPU
    hipError_t err = hipMalloc((void**)&d_matrice, size);
    if (err != hipSuccess) {
        fprintf(stderr, "Erreur d'allocation de mémoire pour d_matrice\n");
        exit(1);
    }
    err = hipMemcpy(d_matrice, s->m, size, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Erreur de copie de mémoire pour d_matrice\n");
        exit(1);
    }

    // Lance le kernel avec le nombre de blocs optimal
    solve_system<<<s->n, s->n+1, sharedSize>>>(d_matrice, s->n);
    hipDeviceSynchronize();

    // Copie les données de retour vers le CPU
    err = hipMemcpy(s->m, d_matrice, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "Erreur de copie de mémoire pour s->m\n");
        exit(1);
    }

    hipFree(d_matrice);

    if (boolAffichage == 1) {
        systeme_afficher(*s);
    }
}

void systeme_remontee_de_gauss(systeme_t* s) {
    for (int i = s->n - 1; i >= 0; i--) {
        double sum = 0;
        for (int j = i + 1; j < s->n; j++) {
            sum += s->matrice[i][j] * s->matrice[j][s->n];
        }
        if (s->matrice[i][i] == 0) {
            s->matrice[i][s->n] = 0;
        } else {
            s->matrice[i][s->n] = (s->matrice[i][s->n] - sum) / s->matrice[i][i];
        }
    }
}

void systeme_output(systeme_t* s, char* pathFile, int boolAffichage) {
    FILE *fichier = fopen(pathFile, "w+");
    if (fichier == NULL) {
        fprintf(stderr, "Erreur: impossible d'ouvrir le fichier\n");
        exit(1);
    }
    fprintf(fichier, "%d\n", s->n);

    systeme_remontee_de_gauss(s);
    for (int i = 0; i < s->n; i++) {
        if (boolAffichage == 1)
            printf("x%d = %.3lf\n", i + 1, s->matrice[i][s->n]);
        fprintf(fichier, "%lf ", s->matrice[i][s->n]);
    }    
    fclose(fichier);
}

void systeme_detruire(systeme_t *s) {
    free(s->m);
    free(s->matrice);
    s->matrice = NULL;
    s->m = NULL;
    s->n = 0;
}
