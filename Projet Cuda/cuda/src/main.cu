#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#include "systeme.h"

void initVariable(int argc, char **argv, char *nameFile, int* boolAffichage) {
    if (argc < 3) {
        fprintf(stderr, "Usage: %s <nom_fichier_input> <boolAffichage>\n", argv[0]);
        exit(1);
    }
    strcpy(nameFile, argv[1]);
    *boolAffichage = atoi(argv[2]);
}

int main(int argc, char** argv) {
    char nameFile[50];
    char pathFile[50] = "./data/";
    char pathFile2[50] = "./data/";
    int boolAffichage = 0;
    initVariable(argc, argv, nameFile, &boolAffichage);

    strcat(pathFile, nameFile);
    strcat(pathFile, ".txt");

    systeme_t s;
    systeme_init(&s);
    systeme_lecture(&s, pathFile);

    hipEvent_t start, stop;
    float milliseconds = 0;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    systeme_resoudre(&s, boolAffichage);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&milliseconds, start, stop);

    strcat(pathFile2, nameFile);
    strcat(pathFile2, "_resultat");
    strcat(pathFile2, ".txt");

    systeme_output(&s, pathFile2, boolAffichage);

    printf("Temps d'execution: %f secondes\n", milliseconds / 1000);

    systeme_detruire(&s);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}