#include "hip/hip_runtime.h"
#include "systeme.h"
#include <ctype.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

void systeme_init(systeme_t *s) {
   s->m = NULL;
   s->matrice = NULL;
   s->n = 0;
   s->nb_pivots = 3;
}

void systeme_lecture(systeme_t *s, char* pathFile) {
    printf("Lecture du fichier %s\n", pathFile);
    FILE *fichier = fopen(pathFile, "r");
    if (fichier == NULL) {
        fprintf(stderr, "Erreur: impossible d'ouvrir le fichier\n");
        exit(1);
    }

    int temp = fscanf(fichier, "%d", &s->n);
    s->m = (double *)malloc((s->n * (s->n + 1)) * sizeof(double));
    if (s->m == NULL) {
        fprintf(stderr, "Erreur: impossible d'allouer la mémoire\n");
        exit(1);
    }

    s->matrice = (double **)malloc(s->n * sizeof(double *));
    if (s->matrice == NULL) {
        fprintf(stderr, "Erreur lors de l'allocation de mémoire pour matrice\n");
        exit(1);
    }

    for (int i = 0; i < s->n; i++) {
        s->matrice[i] = &(s->m[i * (s->n + 1)]);
    }

    for (int i = 0; i < s->n; i++) {
        for (int j = 0; j < (s->n + 1); j++) {
            int temp = fscanf(fichier, "%lf", &s->matrice[i][j]);
        }
    }

    fclose(fichier);
}

void systeme_afficher(systeme_t s) {
    printf("======Affichage du système:======\n");
    for (int i = 0; i < s.n; i++) {
        for (int j = 0; j < s.n + 1; j++) {
            printf("%.3lf ", s.matrice[i][j]);
        }
        printf("\n");
    }
}

void systeme_resoudre(systeme_t *s, int boolAffichage) {
    double** pivots;
    int iter = 0;
    while (iter < s->n-1) {
        if (boolAffichage == 1)
            printf("Iteration %d\n", iter);
        int nb_pivots_to_use = (s->n - iter < s->nb_pivots) ? s->n - iter : s->nb_pivots;
        if (boolAffichage == 1)
            printf("nb_pivots_to_use = %d\n", nb_pivots_to_use);
        if (boolAffichage == 1)
            printf("Iteration %d\n", iter);

        pivots = systeme_choisir_pivot_multiple(s, iter, nb_pivots_to_use, boolAffichage);
        if (pivots == NULL) {
            iter++;
        } else {
            for (int k = 0; k < nb_pivots_to_use; k++) {
                if (pivots[k][iter + k] != 0){
                    pivot_diviser_ligne(pivots[k], iter + k, s->n);
                    if (boolAffichage == 1) {
                        printf("Pivot après division %d:\n", k);
                        afficher_pivots(pivots, nb_pivots_to_use, s->n);
                    }
                    pivot_soustraction_lignes(pivots, k, nb_pivots_to_use, s->n, iter);
                    if (boolAffichage == 1) {
                        printf("Pivot après Soustraction:\n");
                        afficher_pivots(pivots, nb_pivots_to_use, s->n);
                    }
                }
            }
            if (boolAffichage == 1)
                printf("Apres :\n");
            if (boolAffichage == 1)
                afficher_pivots(pivots, nb_pivots_to_use, s->n);
            
            for (int k = 0; k < nb_pivots_to_use; k++) {
                for (int i = 0; i < s->n + 1; i++) {
                    s->matrice[iter + k][i] = pivots[k][i];
                }
            }
            if (boolAffichage == 1) {
                printf("\nMatrice après modif des lignes pivots déjà fait :\n");
                systeme_afficher(*s);
            }
            
            for (int i = iter + nb_pivots_to_use; i < s->n; i++) {
                for (int k = 0; k < nb_pivots_to_use; k++) {
                    double coef = s->matrice[i][iter + k];
                    for (int j = 0; j < s->n + 1; j++) {
                        s->matrice[i][j] -= coef * s->matrice[iter + k][j];
                    }
                }
            }

            if (boolAffichage == 1) {
                printf("\nMatrice après propagation dans dernières lignes:\n");
                systeme_afficher(*s);
            }
            iter += nb_pivots_to_use;
            if (boolAffichage == 1)
                systeme_afficher(*s);
        }
    }
}

double* systeme_choisir_pivot(systeme_t *s, int j) {
    double *pivot = s->matrice[j];
    int indiceDuPivot = j;

    for (int i = j; i < s->n; i++) {
        if (fabs(s->matrice[i][j]) > fabs(pivot[j])) {
            pivot = s->matrice[i];
            indiceDuPivot = i;
        }
    }
    systeme_echanger_lignes(s, j, indiceDuPivot);

    if (pivot[j] == 0) {
        return NULL;
    }

    return pivot;
}

double** systeme_choisir_pivot_multiple(systeme_t *s, int j, int nb_pivots, int boolAffichage) {
    double **pivots = (double **)malloc(s->nb_pivots * sizeof(double *));
    int *indicesDuPivot = (int *)malloc(s->nb_pivots * sizeof(int));

    for (int k = 0; k < nb_pivots; k++) {
        pivots[k] = s->matrice[j + k];
        indicesDuPivot[k] = j + k;

        for (int i = j + k; i < s->n; i++) {
            if (fabs(s->matrice[i][j + k]) > fabs(pivots[k][j + k])) {
                pivots[k] = s->matrice[i];
                indicesDuPivot[k] = i;
            }
        }
        systeme_echanger_lignes(s, j + k, indicesDuPivot[k]);
    }

    if (boolAffichage == 1) {
        printf("Pivots choisis:\n");
        for (int k = 0; k < nb_pivots; k++) {
            printf("Pivot[%d] = %lf : Ligne pivot complete = ", k, pivots[k][j + k]);
            for (int i = 0; i < s->n + 1; i++) {
                printf("%lf ", pivots[k][i]);
            }
            printf("\n");
        }
    }
    free(indicesDuPivot);
    return pivots;
}

void pivot_diviser_ligne(double* row, int k, int n) {
    double pivot = row[k];
    for (int i = k; i < n + 1; i++) {
        row[i] /= pivot;
    }
}

void pivot_soustraction_lignes(double** pivots, int pivot_index, int nb_pivots_to_use, int n, int iter) {
    for (int i = pivot_index + 1; i < nb_pivots_to_use; i++) {
        double coef = pivots[i][pivot_index + iter] / pivots[pivot_index][pivot_index + iter];
        for (int j = iter ; j < n + 1; j++) {
            pivots[i][j] -= coef * pivots[pivot_index][j];
        }
    }
}

void systeme_echanger_lignes(systeme_t *s, int iter, int i) {
    double* temp = s->matrice[iter];
    s->matrice[iter] = s->matrice[i];
    s->matrice[i] = temp;
}

void systeme_remontee_de_gauss(systeme_t *s) {
    for (int i = s->n - 1; i >= 0; i--) {
        double sum = 0;
        for (int j = i + 1; j < s->n; j++) {
            sum += s->matrice[i][j] * s->matrice[j][s->n];
        }
        s->matrice[i][s->n] = (s->matrice[i][s->n] - sum) / s->matrice[i][i];
        s->matrice[i][i] = 1;
        for (int k = 0; k < i; k++) {
            s->matrice[k][s->n] -= s->matrice[k][i] * s->matrice[i][s->n];
            s->matrice[k][i] = 0;
        }
    }
}

void pivot_remontee_de_gauss(double **pivots, int n, int iter_colonne) {
    for (int i = n - 1; i >= 0; i--) {
        double sum = 0;
        for (int j = i + 1; j < n; j++) {
            sum += pivots[i][iter_colonne + j] * pivots[j][iter_colonne + n];
        }
        pivots[i][iter_colonne + n] = (pivots[i][iter_colonne + n] - sum) / pivots[i][iter_colonne + i];
        pivots[i][iter_colonne + i] = 1;
        for (int k = 0; k < i; k++) {
            pivots[k][iter_colonne + n] -= pivots[k][iter_colonne + i] * pivots[i][iter_colonne + n];
            pivots[k][iter_colonne + i] = 0;
        }
    }
}

void afficher_pivots(double** pivots, int nb_pivots, int n) {
    printf("====== Affichage des pivots: ======\n");
    for (int i = 0; i < nb_pivots; i++) {
        for (int j = 0; j < n + 1; j++) {
            printf("%.3lf ", pivots[i][j]);
        }
        printf("\n");
    }
}

void systeme_output(systeme_t *s, char* pathFile, int boolAffichage) {
    FILE *fichier = fopen(pathFile, "w+");
    if (fichier == NULL) {
        fprintf(stderr, "Erreur: impossible d'ouvrir le fichier\n");
        exit(1);
    }
    fprintf(fichier, "%d\n", s->n);

    systeme_remontee_de_gauss(s);
    for (int i = 0; i < s->n; i++) {
       
        if (boolAffichage) printf("x%d = %.3lf\n", i + 1, s->matrice[i][s->n]);
        fprintf(fichier, "%lf ", s->matrice[i][s->n]);
    }
    fclose(fichier);
}

void systeme_detruire(systeme_t *s) {
    free(s->m);
    free(s->matrice);
    s->matrice = NULL;
    s->m = NULL;
    s->n = 0;
}
