#include "hip/hip_runtime.h"
#include "systeme.h"
#include <ctype.h>

void systeme_init(systeme_t *s) {
   s->m = NULL;
   s->matrice = NULL;
   s->n = 0;
   s->nb_pivots = 3;
}

void systeme_lecture(systeme_t *s, char* pathFile) {
    printf("Lecture du fichier %s\n", pathFile);
    FILE *fichier = fopen(pathFile, "r");
    if (fichier == NULL) {
        fprintf(stderr, "Erreur: impossible d'ouvrir le fichier\n");
        exit(1);
    }

    int temp = fscanf(fichier, "%d", &s->n);
    s->m = (double *)malloc((s->n * (s->n + 1)) * sizeof(double));
    if (s->m == NULL) {
        fprintf(stderr, "Erreur: impossible d'allouer la mémoire\n");
        exit(1);
    }

    s->matrice = (double **)malloc(s->n * sizeof(double *));
    if (s->matrice == NULL) {
        fprintf(stderr, "Erreur lors de l'allocation de mémoire pour matrice\n");
        exit(1);
    }

    for (int i = 0; i < s->n; i++) {
        s->matrice[i] = &(s->m[i * (s->n + 1)]);
    }

    for (int i = 0; i < s->n; i++) {
        for (int j = 0; j < (s->n + 1); j++) {
            int temp = fscanf(fichier, "%lf", &s->matrice[i][j]);
        }
    }

    fclose(fichier);
}

void systeme_afficher(systeme_t s) {
    printf("======Affichage du système:======\n");
    for (int i = 0; i < s.n; i++) {
        for (int j = 0; j < s.n + 1; j++) {
            printf("%.3lf ", s.matrice[i][j]);
        }
        printf("\n");
    }
}

void systeme_resoudre(systeme_t *s, int boolAffichage) {
    double** pivots;
    int iter = 0;
    while (iter < s->n-1) {
        if (boolAffichage == 1)
            printf("Iteration %d\n", iter);
        int nb_pivots_to_use = (s->n - iter < s->nb_pivots) ? s->n - iter : s->nb_pivots;
        if (boolAffichage == 1)
            printf("nb_pivots_to_use = %d\n", nb_pivots_to_use);
        if (boolAffichage == 1)
            printf("Iteration %d\n", iter);

        pivots = systeme_choisir_pivot_multiple(s, iter, nb_pivots_to_use, boolAffichage);
        if (pivots == NULL) {
            iter++;
        } else {
            for (int k = 0; k < nb_pivots_to_use; k++) {
                if (pivots[k][iter + k] != 0){
                    pivot_diviser_ligne(pivots[k], iter + k, s->n);
                    if (boolAffichage == 1) {
                        printf("Pivot après division %d:\n", k);
                        afficher_pivots(pivots, nb_pivots_to_use, s->n);
                    }
                    pivot_soustraction_lignes(pivots, k, nb_pivots_to_use, s->n, iter);
                    if (boolAffichage == 1) {
                        printf("Pivot après Soustraction:\n");
                        afficher_pivots(pivots, nb_pivots_to_use, s->n);
                    }
                }
            }
            if (boolAffichage == 1)
                printf("Apres :\n");
            if (boolAffichage == 1)
                afficher_pivots(pivots, nb_pivots_to_use, s->n);
            
            for (int k = 0; k < nb_pivots_to_use; k++) {
                for (int i = 0; i < s->n + 1; i++) {
                    s->matrice[iter + k][i] = pivots[k][i];
                }
            }
            if (boolAffichage == 1) {
                printf("\nMatrice après modif des lignes pivots déjà fait :\n");
                systeme_afficher(*s);
            }
            
            for (int i = iter + nb_pivots_to_use; i < s->n; i++) {
                for (int k = 0; k < nb_pivots_to_use; k++) {
                    double coef = s->matrice[i][iter + k];
                    for (int j = 0; j < s->n + 1; j++) {
                        s->matrice[i][j] -= coef * s->matrice[iter + k][j];
                    }
                }
            }

            if (boolAffichage == 1) {
                printf("\nMatrice après propagation dans dernières lignes:\n");
                systeme_afficher(*s);
            }
            iter += nb_pivots_to_use;
            if (boolAffichage == 1)
                systeme_afficher(*s);
        }
    }
}

double* systeme_choisir_pivot(systeme_t *s, int j) {
    double *pivot = s->matrice[j];
    int indiceDuPivot = j;

    for (int i = j; i < s->n; i++) {
        if (fabs(s->matrice[i][j]) > fabs(pivot[j])) {
            pivot = s->matrice[i];
            indiceDuPivot = i;
        }
    }
    systeme_echanger_lignes(s, j, indiceDuPivot);

    if (pivot[j] == 0) {
        return NULL;
    }

    return pivot;
}


double** systeme_choisir_pivot_multiple(systeme_t *s, int j, int nb_pivots, int boolAffichage) {
    double **pivots = (double **)malloc(s->nb_pivots * sizeof(double *));
    int *indicesDuPivot = (int *)malloc(s->nb_pivots * sizeof(int));

    for (int k = 0; k < nb_pivots; k++) {
        pivots[k] = s->matrice[j + k];
        indicesDuPivot[k] = j + k;

        for (int i = j + k; i < s->n; i++) {
            if (fabs(s->matrice[i][j + k]) > fabs(pivots[k][j + k])) {
                pivots[k] = s->matrice[i];
                indicesDuPivot[k] = i;
            }
        }
        systeme_echanger_lignes(s, j + k, indicesDuPivot[k]);
    }

    if (boolAffichage == 1) {
        printf("Pivots choisis:\n");
        for (int k = 0; k < nb_pivots; k++) {
            printf("Pivot[%d] = %lf : Ligne pivot complete = ", k, pivots[k][j + k]);
            for (int i = 0; i < s->n + 1; i++) {
                printf("%lf ", pivots[k][i]);
            }
            printf("\n");
        }
    }
    free(indicesDuPivot);
    return pivots;
}


/*
double** systeme_choisir_pivot_multiple(systeme_t *s, int j, int nb_pivots, int boolAffichage) {
    double **pivots = (double **)malloc(nb_pivots * sizeof(double *));
    int *indicesDuPivot = (int *)malloc(nb_pivots * sizeof(int));
    int *lignes_utilisees = (int *)calloc(s->n, sizeof(int)); // Tableau pour marquer les lignes utilisées

    // Allouer la mémoire sur le GPU pour les indices, les lignes utilisées et la matrice
    int *d_indicesDuPivot, *d_lignes_utilisees;
    double *d_matrice;
    hipMalloc((void**)&d_indicesDuPivot, nb_pivots * sizeof(int));
    hipMalloc((void**)&d_lignes_utilisees, s->n * sizeof(int));
    hipMalloc((void**)&d_matrice, s->n * (s->n + 1) * sizeof(double));

    // Copier les données de la matrice et du tableau des lignes utilisées sur le GPU
    hipMemcpy(d_matrice, s->m, s->n * (s->n + 1) * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_lignes_utilisees, lignes_utilisees, s->n * sizeof(int), hipMemcpyHostToDevice);

    // Lancer le noyau CUDA pour trouver les pivots
    int blockSize = 256;
    int numBlocks = (nb_pivots + blockSize - 1) / blockSize;
    trouver_pivots<<<numBlocks, blockSize>>>(d_matrice, d_indicesDuPivot, d_lignes_utilisees, s->n, j, nb_pivots);

    // Copier les indices de pivots trouvés depuis le GPU vers le CPU
    hipMemcpy(indicesDuPivot, d_indicesDuPivot, nb_pivots * sizeof(int), hipMemcpyDeviceToHost);

    // Libérer la mémoire GPU
    hipFree(d_indicesDuPivot);
    hipFree(d_lignes_utilisees);
    hipFree(d_matrice);

    // Organiser les lignes du système en fonction des pivots trouvés
    for (int k = 0; k < nb_pivots; k++) {
        if (indicesDuPivot[k] != -1) {
            systeme_echanger_lignes(s, j + k, indicesDuPivot[k]);
            pivots[k] = s->matrice[j + k];
        } else {
            pivots[k] = NULL; // Si aucun pivot n'a été trouvé pour cette colonne
        }
    }

    if (boolAffichage == 1) {
        printf("Pivots choisis:\n");
        for (int k = 0; k < nb_pivots; k++) {
            if (pivots[k] != NULL) {
                printf("Pivot[%d] = %lf : Ligne pivot complete = ", k, pivots[k][j + k]);
                for (int i = 0; i < s->n + 1; i++) {
                    printf("%lf ", pivots[k][i]);
                }
                printf("\n");
            } else {
                printf("Pivot[%d] = Aucun pivot trouvé\n", k);
            }
        }
    }

    free(indicesDuPivot);
    free(lignes_utilisees);
    return pivots;
}
__global__ void trouver_pivots(double* matrice, int* indicesDuPivot, int* lignes_utilisees, int n, int k, int nb_pivots) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < nb_pivots) {
        int col = k + idx;
        int max_idx = -1;
        double max_val = -1.0;
        for (int i = k; i < n; ++i) {
            if (!lignes_utilisees[i]) {
                double val = fabs(matrice[i * (n + 1) + col]);
                if (val > max_val) {
                    max_val = val;
                    max_idx = i;
                }
            }
        }
        if (max_idx != -1) {
            indicesDuPivot[idx] = max_idx;
            lignes_utilisees[max_idx] = 1; // Marquer cette ligne comme utilisée
        } else {
            indicesDuPivot[idx] = -1; // Indiquer qu'aucun pivot n'a été trouvé pour cette colonne
        }
    }
}*/

__global__ void divide_row(double* row, int k, int n) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= k && idx < n + 1) {
        row[idx] /= row[k];
    }
}

__global__ void subtract_rows(double* pivots, int pivot_index, int nb_pivots_to_use, int n, int iter, int stride) {
    int i = blockIdx.x;
    int j = threadIdx.x + iter;
    
    if (i > pivot_index && i < nb_pivots_to_use && j < n + 1) {
        double coef = pivots[i * stride + pivot_index + iter] / pivots[pivot_index * stride + pivot_index + iter];
        pivots[i * stride + j] -= coef * pivots[pivot_index * stride + j];
    }
}

void pivot_diviser_ligne(double* row, int k, int n) {
    int size = n + 1;
    double* d_row;
    hipMalloc((void**)&d_row, size * sizeof(double));
    hipMemcpy(d_row, row, size * sizeof(double), hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (size + blockSize - 1) / blockSize;
    divide_row<<<numBlocks, blockSize>>>(d_row, k, n);

    hipMemcpy(row, d_row, size * sizeof(double), hipMemcpyDeviceToHost);
    hipFree(d_row);
}

void pivot_soustraction_lignes(double** pivots, int pivot_index, int nb_pivots_to_use, int n, int iter) {
    int size = nb_pivots_to_use * (n + 1);
    double* d_pivots;
    hipMalloc((void**)&d_pivots, size * sizeof(double));
    for (int i = 0; i < nb_pivots_to_use; i++) {
        hipMemcpy(d_pivots + i * (n + 1), pivots[i], (n + 1) * sizeof(double), hipMemcpyHostToDevice);
    }

    int blockSize = 256;
    int numBlocks = nb_pivots_to_use;
    subtract_rows<<<numBlocks, blockSize>>>(d_pivots, pivot_index, nb_pivots_to_use, n, iter, n + 1);

    for (int i = 0; i < nb_pivots_to_use; i++) {
        hipMemcpy(pivots[i], d_pivots + i * (n + 1), (n + 1) * sizeof(double), hipMemcpyDeviceToHost);
    }
    hipFree(d_pivots);
}

void systeme_echanger_lignes(systeme_t *s, int iter, int i) {
    double* temp = s->matrice[iter];
    s->matrice[iter] = s->matrice[i];
    s->matrice[i] = temp;
}

void systeme_remontee_de_gauss(systeme_t *s) {
    for (int i = s->n - 1; i >= 0; i--) {
        double sum = 0;
        for (int j = i + 1; j < s->n; j++) {
            sum += s->matrice[i][j] * s->matrice[j][s->n];
        }
        s->matrice[i][s->n] = (s->matrice[i][s->n] - sum) / s->matrice[i][i];
        s->matrice[i][i] = 1;
        for (int k = 0; k < i; k++) {
            s->matrice[k][s->n] -= s->matrice[k][i] * s->matrice[i][s->n];
            s->matrice[k][i] = 0;
        }
    }
}

void pivot_remontee_de_gauss(double **pivots, int n, int iter_colonne) {
    for (int i = n - 1; i >= 0; i--) {
        double sum = 0;
        for (int j = i + 1; j < n; j++) {
            sum += pivots[i][iter_colonne + j] * pivots[j][iter_colonne + n];
        }
        pivots[i][iter_colonne + n] = (pivots[i][iter_colonne + n] - sum) / pivots[i][iter_colonne + i];
        pivots[i][iter_colonne + i] = 1;
        for (int k = 0; k < i; k++) {
            pivots[k][iter_colonne + n] -= pivots[k][iter_colonne + i] * pivots[i][iter_colonne + n];
            pivots[k][iter_colonne + i] = 0;
        }
    }
}

void afficher_pivots(double** pivots, int nb_pivots, int n) {
    printf("====== Affichage des pivots: ======\n");
    for (int i = 0; i < nb_pivots; i++) {
        for (int j = 0; j < n + 1; j++) {
            printf("%.3lf ", pivots[i][j]);
        }
        printf("\n");
    }
}

void systeme_output(systeme_t *s, char* pathFile, int boolAffichage) {
    FILE *fichier = fopen(pathFile, "w+");
    if (fichier == NULL) {
        fprintf(stderr, "Erreur: impossible d'ouvrir le fichier\n");
        exit(1);
    }
    fprintf(fichier, "%d\n", s->n);

    systeme_remontee_de_gauss(s);
    for (int i = 0; i < s->n; i++) {
       
        if (boolAffichage) printf("x%d = %.3lf\n", i + 1, s->matrice[i][s->n]);
        fprintf(fichier, "%lf ", s->matrice[i][s->n]);
    }
    fclose(fichier);
}

void systeme_detruire(systeme_t *s) {
    free(s->m);
    free(s->matrice);
    s->matrice = NULL;
    s->m = NULL;
    s->n = 0;
}
